#include "hip/hip_runtime.h"
#include "rope_kernel.cuh"
#include <cstdint>

// Rope operation before MHA
// usage:
//  thread = 128
//  block = (dim + threads - 1) / threads;

__device__ void rope_calc(float fcr, float fci, float* vec, int32_t idx) {
  float2* vec_ptr = reinterpret_cast<float2*>(vec + idx);
  float2 vec_value = *vec_ptr;
  *vec_ptr =
      make_float2(vec_value.x * fcr - vec_value.y * fci, vec_value.x * fci + vec_value.y * fcr);
}

// wrap this kernel
//  sin_cache: [head_size, seq_len]
//  cos_cache: [head_size, seq_len]
__global__ void rope_kernel_cu_fp32(int pos, int dim, int kv_dim, int head_size,
                                    const float* input_q, const float* input_k,
                                    const float* sin_cache, const float* cos_cache) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  idx = idx * 2;
  if (idx >= dim) {
    return;
  }

  int head_dim = idx % head_size;
  float fci = *(sin_cache + pos * head_size + head_dim);
  float fcr = *(cos_cache + pos * head_size + head_dim);

  rope_calc(fcr, fci, const_cast<float*>(input_q), idx);
  if (idx >= kv_dim) {
    return;
  }
  rope_calc(fcr, fci, const_cast<float*>(input_k), idx);
}

// warp this kernel

// function:
//   position embedding calculation
//   for a certain dim D in one head, generate two [max_seq_len] sized position feature
//   sin: [sin(0 * (1 / 10000^(D / 64))), sin(1 * (1 / 10000^(D / 64))), ..., sin((max_seq_len-1) * (1 / 10000^(D / 64)))]
//   cos: [cos(0 * (1 / 10000^(D / 64))), cos(1 * (1 / 10000^(D / 64))), ..., cos((max_seq_len-1) * (1 / 10000^(D / 64)))]
// usage:
//   block size is the same as your head size, typically 32
__global__ void sin_cos_calc(int head_size, int max_seq_len, float* sin_cache, float* cos_cache) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  int head_dim = idx % head_size;
  for (int pos = 0; pos < max_seq_len; ++pos) {
    float freq = 1.0f / pow(10000.0f, static_cast<float>(head_dim) / static_cast<float>(head_size));
    float val = static_cast<float>(pos) * freq;
    float fcr = cosf(val);
    float fci = sinf(val);
    *(sin_cache + pos * head_size + head_dim) = fci;
    *(cos_cache + pos * head_size + head_dim) = fcr;
  }
}

void sin_cos_cache_calc_cu(int head_size, int max_seq_len, const tensor::Tensor& sin_cache,
                           const tensor::Tensor& cos_cache, hipStream_t stream) {
  CHECK_EQ(sin_cache.is_empty(), false);
  CHECK_EQ(cos_cache.is_empty(), false);
  int threads = head_size;
  if (stream) {
    sin_cos_calc<<<1, threads, 0, stream>>>(head_size, max_seq_len,
                                            const_cast<float*>(sin_cache.ptr<float>()),
                                            const_cast<float*>(cos_cache.ptr<float>()));
  } else {
    sin_cos_calc<<<1, threads>>>(head_size, max_seq_len, const_cast<float*>(sin_cache.ptr<float>()),
                                 const_cast<float*>(cos_cache.ptr<float>()));
  }
}
