#include "hip/hip_runtime.h"
#include <cub/block/block_reduce.cuh>
#include <cstdint>

// warp this kernel
// perform gemv for QKV and MLP layer
// usage:
//   shape of weight of linear layer: [M, N]
//   block size = 128
//   grid size = M  (row num of weight)
template <int THREAD_PER_BLOCK, int ROW_PER_BLOCK>
__global__ void matmul_kernel_cu_fp32(const float* input, const float* weight, float* output, int M,
                                      int N) {
  __shared__ float sdata[THREAD_PER_BLOCK];
  unsigned int tid = threadIdx.x;

  int start_row = blockIdx.x * ROW_PER_BLOCK;
  int end_row = start_row + ROW_PER_BLOCK;
  if (start_row >= N) {
    return;
  }

  constexpr int pack_size = 4;
  const int pack_num = M / pack_size;
  const int pack_off = pack_size * pack_num;

#pragma unroll
  for (int p = start_row; p < end_row; ++p) {
    sdata[tid] = 0;
    int row_offset = p * M;
    float4* input_float4_ptr = (float4*)input;
    float4* weight_float4_ptr = (float4*)(weight + row_offset);

#pragma unroll
    for (int i = tid; i < pack_num; i += blockDim.x) {
      float4 input_float4 = *(input_float4_ptr + i);
      float4 weight_float4 = *(weight_float4_ptr + i);
      float part_sum = input_float4.x * weight_float4.x + input_float4.y * weight_float4.y +
                       input_float4.z * weight_float4.z + input_float4.w * weight_float4.w;
      sdata[tid] += part_sum;
    }

    for (int i = pack_off + tid; i < M; i += blockDim.x) {
      sdata[tid] += input[i] * weight[row_offset + i];
    }

    __syncthreads();

    using BlockReduce = hipcub::BlockReduce<float, THREAD_PER_BLOCK>;
    __shared__ typename BlockReduce::TempStorage temp;
    float part_sum = BlockReduce(temp).Sum(sdata[tid]);
    __syncthreads();

    if (tid == 0) {
      output[p] = part_sum;
    }
    __syncthreads();
  }
}
